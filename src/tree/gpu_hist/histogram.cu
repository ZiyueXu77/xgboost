#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2024, XGBoost Contributors
 */
#include <thrust/iterator/transform_iterator.h>  // for make_transform_iterator

#include <algorithm>
#include <cstdint>  // uint32_t, int32_t

#include "../../collective/aggregator.h"
#include "../../common/deterministic.cuh"
#include "../../common/device_helpers.cuh"
#include "../../data/ellpack_page.cuh"
#include "histogram.cuh"
#include "row_partitioner.cuh"
#include "xgboost/base.h"

#include "../../common/device_helpers.cuh"
#if defined(XGBOOST_USE_FEDERATED)
#include "../../../plugin/federated/federated_hist.h"  // for FederataedHistPolicy
#else
#include "../../common/error_msg.h"  // for NoFederated
#endif

namespace xgboost::tree {
namespace {
struct Pair {
  GradientPair first;
  GradientPair second;
};
__host__ XGBOOST_DEV_INLINE Pair operator+(Pair const& lhs, Pair const& rhs) {
  return {lhs.first + rhs.first, lhs.second + rhs.second};
}
}  // anonymous namespace

struct Clip : public thrust::unary_function<GradientPair, Pair> {
  static XGBOOST_DEV_INLINE float Pclip(float v) { return v > 0 ? v : 0; }
  static XGBOOST_DEV_INLINE float Nclip(float v) { return v < 0 ? abs(v) : 0; }

  XGBOOST_DEV_INLINE Pair operator()(GradientPair x) const {
    auto pg = Pclip(x.GetGrad());
    auto ph = Pclip(x.GetHess());

    auto ng = Nclip(x.GetGrad());
    auto nh = Nclip(x.GetHess());

    return {GradientPair{pg, ph}, GradientPair{ng, nh}};
  }
};

/**
 * In algorithm 5 (see common::CreateRoundingFactor) the bound is calculated as
 * $max(|v_i|) * n$.  Here we use the bound:
 *
 * \begin{equation}
 *   max( fl(\sum^{V}_{v_i>0}{v_i}), fl(\sum^{V}_{v_i<0}|v_i|) )
 * \end{equation}
 *
 * to avoid outliers, as the full reduction is reproducible on GPU with reduction tree.
 */
GradientQuantiser::GradientQuantiser(Context const* ctx, common::Span<GradientPair const> gpair,
                                     MetaInfo const& info) {
  using GradientSumT = GradientPairPrecise;
  using T = typename GradientSumT::ValueT;
  dh::XGBCachingDeviceAllocator<char> alloc;

  thrust::device_ptr<GradientPair const> gpair_beg{gpair.data()};
  auto beg = thrust::make_transform_iterator(gpair_beg, Clip());
  Pair p =
      dh::Reduce(thrust::cuda::par(alloc), beg, beg + gpair.size(), Pair{}, thrust::plus<Pair>{});
  // Treat pair as array of 4 primitive types to allreduce
  using ReduceT = typename decltype(p.first)::ValueT;
  static_assert(sizeof(Pair) == sizeof(ReduceT) * 4, "Expected to reduce four elements.");
  auto rc = collective::GlobalSum(ctx, info, linalg::MakeVec(reinterpret_cast<ReduceT*>(&p), 4));
  collective::SafeColl(rc);

  GradientPair positive_sum{p.first}, negative_sum{p.second};

  std::size_t total_rows = gpair.size();
  rc = collective::GlobalSum(ctx, info, linalg::MakeVec(&total_rows, 1));
  collective::SafeColl(rc);

  auto histogram_rounding =
      GradientSumT{common::CreateRoundingFactor<T>(
                       std::max(positive_sum.GetGrad(), negative_sum.GetGrad()), total_rows),
                   common::CreateRoundingFactor<T>(
                       std::max(positive_sum.GetHess(), negative_sum.GetHess()), total_rows)};

  using IntT = typename GradientPairInt64::ValueT;

  /**
   * Factor for converting gradients from fixed-point to floating-point.
   */
  to_floating_point_ =
      histogram_rounding /
      static_cast<T>(static_cast<IntT>(1)
                     << (sizeof(typename GradientSumT::ValueT) * 8 - 2));  // keep 1 for sign bit
  /**
   * Factor for converting gradients from floating-point to fixed-point. For
   * f64:
   *
   *   Precision = 64 - 1 - log2(rounding)
   *
   * rounding is calcuated as exp(m), see the rounding factor calcuation for
   * details.
   */
  to_fixed_point_ = GradientSumT(static_cast<T>(1) / to_floating_point_.GetGrad(),
                                 static_cast<T>(1) / to_floating_point_.GetHess());
}

XGBOOST_DEV_INLINE void AtomicAddGpairShared(xgboost::GradientPairInt64* dest,
                                             xgboost::GradientPairInt64 const& gpair) {
  auto dst_ptr = reinterpret_cast<int64_t *>(dest);
  auto g = gpair.GetQuantisedGrad();
  auto h = gpair.GetQuantisedHess();

  AtomicAdd64As32(dst_ptr, g);
  AtomicAdd64As32(dst_ptr + 1, h);
}

// Global 64 bit integer atomics at the time of writing do not benefit from being separated into two
// 32 bit atomics
XGBOOST_DEV_INLINE void AtomicAddGpairGlobal(xgboost::GradientPairInt64* dest,
                                             xgboost::GradientPairInt64 const& gpair) {
  auto dst_ptr = reinterpret_cast<uint64_t*>(dest);
  auto g = gpair.GetQuantisedGrad();
  auto h = gpair.GetQuantisedHess();

  atomicAdd(dst_ptr,
            *reinterpret_cast<uint64_t*>(&g));
  atomicAdd(dst_ptr + 1,
            *reinterpret_cast<uint64_t*>(&h));
}

template <int kBlockThreads, int kItemsPerThread,
          int kItemsPerTile = kBlockThreads * kItemsPerThread>
class HistogramAgent {
  GradientPairInt64* smem_arr_;
  GradientPairInt64* d_node_hist_;
  using Idx = RowPartitioner::RowIndexT;

  dh::LDGIterator<const Idx> d_ridx_;
  const GradientPair* d_gpair_;
  const FeatureGroup group_;
  const EllpackDeviceAccessor& matrix_;
  const int feature_stride_;
  const std::size_t n_elements_;
  const GradientQuantiser& rounding_;

 public:
  __device__ HistogramAgent(GradientPairInt64* smem_arr,
                            GradientPairInt64* __restrict__ d_node_hist, const FeatureGroup& group,
                            const EllpackDeviceAccessor& matrix, common::Span<const Idx> d_ridx,
                            const GradientQuantiser& rounding, const GradientPair* d_gpair)
      : smem_arr_(smem_arr),
        d_node_hist_(d_node_hist),
        d_ridx_(d_ridx.data()),
        group_(group),
        matrix_(matrix),
        feature_stride_(matrix.is_dense ? group.num_features : matrix.row_stride),
        n_elements_(feature_stride_ * d_ridx.size()),
        rounding_(rounding),
        d_gpair_(d_gpair) {}

  __device__ void ProcessPartialTileShared(std::size_t offset) {
    for (std::size_t idx = offset + threadIdx.x;
         idx < std::min(offset + kBlockThreads * kItemsPerTile, n_elements_);
         idx += kBlockThreads) {
      Idx ridx = d_ridx_[idx / feature_stride_];
      Idx midx = (ridx - matrix_.base_rowid) * matrix_.row_stride + group_.start_feature +
                  idx % feature_stride_;
      bst_bin_t gidx = matrix_.gidx_iter[midx] - group_.start_bin;
      if (matrix_.is_dense || gidx != matrix_.NumBins()) {
        auto adjusted = rounding_.ToFixedPoint(d_gpair_[ridx]);
        AtomicAddGpairShared(smem_arr_ + gidx, adjusted);
      }
    }
  }
  // Instruction level parallelism by loop unrolling
  // Allows the kernel to pipeline many operations while waiting for global memory
  // Increases the throughput of this kernel significantly
  __device__ void ProcessFullTileShared(std::size_t offset) {
    std::size_t idx[kItemsPerThread];
    int ridx[kItemsPerThread];
    int gidx[kItemsPerThread];
    GradientPair gpair[kItemsPerThread];
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      idx[i] = offset + i * kBlockThreads + threadIdx.x;
    }
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      ridx[i] = d_ridx_[idx[i] / feature_stride_];
    }
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      gpair[i] = d_gpair_[ridx[i]];
      gidx[i] = matrix_.gidx_iter[(ridx[i] - matrix_.base_rowid) * matrix_.row_stride +
                                  group_.start_feature + idx[i] % feature_stride_];
    }
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      if ((matrix_.is_dense || gidx[i] != matrix_.NumBins())) {
        auto adjusted = rounding_.ToFixedPoint(gpair[i]);
        AtomicAddGpairShared(smem_arr_ + gidx[i] - group_.start_bin, adjusted);
      }
    }
  }
  __device__ void BuildHistogramWithShared() {
    dh::BlockFill(smem_arr_, group_.num_bins, GradientPairInt64{});
    __syncthreads();

    std::size_t offset = blockIdx.x * kItemsPerTile;
    while (offset + kItemsPerTile <= n_elements_) {
      ProcessFullTileShared(offset);
      offset += kItemsPerTile * gridDim.x;
    }
    ProcessPartialTileShared(offset);

    // Write shared memory back to global memory
    __syncthreads();
    for (auto i : dh::BlockStrideRange(0, group_.num_bins)) {
      AtomicAddGpairGlobal(d_node_hist_ + group_.start_bin + i, smem_arr_[i]);
    }
  }

  __device__ void BuildHistogramWithGlobal() {
    for (auto idx : dh::GridStrideRange(static_cast<std::size_t>(0), n_elements_)) {
      Idx ridx = d_ridx_[idx / feature_stride_];
      bst_bin_t gidx = matrix_.gidx_iter[(ridx - matrix_.base_rowid) * matrix_.row_stride +
                                         group_.start_feature + idx % feature_stride_];
      if (matrix_.is_dense || gidx != matrix_.NumBins()) {
        auto adjusted = rounding_.ToFixedPoint(d_gpair_[ridx]);
        AtomicAddGpairGlobal(d_node_hist_ + gidx, adjusted);
      }
    }
  }
};

template <bool use_shared_memory_histograms, int kBlockThreads, int kItemsPerThread>
__global__ void __launch_bounds__(kBlockThreads)
    SharedMemHistKernel(const EllpackDeviceAccessor matrix,
                        const FeatureGroupsAccessor feature_groups,
                        common::Span<const RowPartitioner::RowIndexT> d_ridx,
                        GradientPairInt64* __restrict__ d_node_hist,
                        const GradientPair* __restrict__ d_gpair,
                        GradientQuantiser const rounding) {
  extern __shared__ char smem[];
  const FeatureGroup group = feature_groups[blockIdx.y];
  auto smem_arr = reinterpret_cast<GradientPairInt64*>(smem);
  auto agent = HistogramAgent<kBlockThreads, kItemsPerThread>(smem_arr, d_node_hist, group, matrix,
                                                              d_ridx, rounding, d_gpair);
  if (use_shared_memory_histograms) {
    agent.BuildHistogramWithShared();
  } else {
    agent.BuildHistogramWithGlobal();
  }
}

namespace {
constexpr std::int32_t kBlockThreads = 1024;
constexpr std::int32_t kItemsPerThread = 8;
constexpr std::int32_t ItemsPerTile() { return kBlockThreads * kItemsPerThread; }
}  // namespace

// Use auto deduction guide to workaround compiler error.
template <auto Global = SharedMemHistKernel<false, kBlockThreads, kItemsPerThread>,
          auto Shared = SharedMemHistKernel<true, kBlockThreads, kItemsPerThread>>
struct HistogramKernel {
  decltype(Global) global_kernel{SharedMemHistKernel<false, kBlockThreads, kItemsPerThread>};
  decltype(Shared) shared_kernel{SharedMemHistKernel<true, kBlockThreads, kItemsPerThread>};
  bool shared{false};
  std::uint32_t grid_size{0};
  std::size_t smem_size{0};

  HistogramKernel(Context const* ctx, FeatureGroupsAccessor const& feature_groups,
                  bool force_global_memory) {
    // Decide whether to use shared memory
    // Opt into maximum shared memory for the kernel if necessary
    std::size_t max_shared_memory = dh::MaxSharedMemoryOptin(ctx->Ordinal());

    this->smem_size = sizeof(GradientPairInt64) * feature_groups.max_group_bins;
    this->shared = !force_global_memory && smem_size <= max_shared_memory;
    this->smem_size = this->shared ? this->smem_size : 0;

    auto init = [&](auto& kernel) {
      if (this->shared) {
        dh::safe_cuda(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                                           max_shared_memory));
      }

      // determine the launch configuration
      std::int32_t num_groups = feature_groups.NumGroups();
      std::int32_t n_mps = 0;
      dh::safe_cuda(hipDeviceGetAttribute(&n_mps, hipDeviceAttributeMultiprocessorCount, ctx->Ordinal()));

      std::int32_t n_blocks_per_mp = 0;
      dh::safe_cuda(hipOccupancyMaxActiveBlocksPerMultiprocessor(&n_blocks_per_mp, kernel,
                                                                  kBlockThreads, this->smem_size));

      // This gives the number of blocks to keep the device occupied Use this as the
      // maximum number of blocks
      this->grid_size = n_blocks_per_mp * n_mps;
    };

    init(this->global_kernel);
    init(this->shared_kernel);
  }
};

class DeviceHistogramBuilderImpl {
  std::unique_ptr<HistogramKernel<>> kernel_{nullptr};
  bool force_global_memory_{false};

 public:
  void Reset(Context const* ctx, FeatureGroupsAccessor const& feature_groups,
             bool force_global_memory) {
    this->kernel_ = std::make_unique<HistogramKernel<>>(ctx, feature_groups, force_global_memory);
    this->force_global_memory_ = force_global_memory;
  }

  void BuildHistogram(HIPContext const* ctx, EllpackDeviceAccessor const& matrix,
                      FeatureGroupsAccessor const& feature_groups,
                      common::Span<GradientPair const> gpair,
                      common::Span<const std::uint32_t> d_ridx,
                      common::Span<GradientPairInt64> histogram, GradientQuantiser rounding) {
    CHECK(kernel_);
    // Otherwise launch blocks such that each block has a minimum amount of work to do
    // There are fixed costs to launching each block, e.g. zeroing shared memory
    // The below amount of minimum work was found by experimentation
    int columns_per_group = common::DivRoundUp(matrix.row_stride, feature_groups.NumGroups());
    // Average number of matrix elements processed by each group
    std::size_t items_per_group = d_ridx.size() * columns_per_group;

    // Allocate number of blocks such that each block has about kMinItemsPerBlock work
    // Up to a maximum where the device is saturated
    auto constexpr kMinItemsPerBlock = ItemsPerTile();
    auto grid_size = std::min(kernel_->grid_size, static_cast<std::uint32_t>(common::DivRoundUp(
                                                      items_per_group, kMinItemsPerBlock)));

    if (this->force_global_memory_ || !this->kernel_->shared) {
      dh::LaunchKernel{dim3(grid_size, feature_groups.NumGroups()),  // NOLINT
                       static_cast<uint32_t>(kBlockThreads), kernel_->smem_size,
                       ctx->Stream()}(kernel_->global_kernel, matrix, feature_groups, d_ridx,
                                      histogram.data(), gpair.data(), rounding);
    } else {
      dh::LaunchKernel{dim3(grid_size, feature_groups.NumGroups()),  // NOLINT
                       static_cast<uint32_t>(kBlockThreads), kernel_->smem_size,
                       ctx->Stream()}(kernel_->shared_kernel, matrix, feature_groups, d_ridx,
                                      histogram.data(), gpair.data(), rounding);
    }
  }
};

DeviceHistogramBuilder::DeviceHistogramBuilder()
    : p_impl_{std::make_unique<DeviceHistogramBuilderImpl>()} {}

DeviceHistogramBuilder::~DeviceHistogramBuilder() = default;

void DeviceHistogramBuilder::Reset(Context const* ctx, FeatureGroupsAccessor const& feature_groups,
                                   bool force_global_memory) {
  this->p_impl_->Reset(ctx, feature_groups, force_global_memory);
}

void DeviceHistogramBuilder::BuildHistogram(Context const* ctx,
                                            EllpackDeviceAccessor const& matrix,
                                            FeatureGroupsAccessor const& feature_groups,
                                            common::Span<GradientPair const> gpair,
                                            common::Span<const std::uint32_t> ridx,
                                            common::Span<GradientPairInt64> histogram,
                                            GradientQuantiser rounding, MetaInfo const& info) {

  auto IsSecureVertical = !info.IsRowSplit() && collective::IsDistributed() && collective::IsEncrypted();
  if (!IsSecureVertical) {
    // Regular training, build histogram locally
    this->p_impl_->BuildHistogram(ctx->CUDACtx(), matrix, feature_groups, gpair, ridx, histogram, rounding);
  } else {
    // Encrypted vertical, build histogram using federated plugin
    auto const &comm = collective::GlobalCommGroup()->Ctx(ctx, DeviceOrd::CPU());
    auto const &fed = dynamic_cast<collective::FederatedComm const &>(comm);
    auto plugin = fed.EncryptionPlugin();
    // Transmit matrix to plugin
    if(!is_aggr_context_initialized_){
      std::cout << "Initialized Plugin Context" << std::endl;
      // Get cutptrs
      std::vector<uint32_t> h_cuts_ptr(matrix.feature_segments.size());
      dh::CopyDeviceSpanToVector(&h_cuts_ptr, matrix.feature_segments);
      common::Span<std::uint32_t const> cutptrs = common::Span<std::uint32_t const>(h_cuts_ptr.data(), h_cuts_ptr.size());
      std::cout << "cutptrs.size() = " << h_cuts_ptr.size() << std::endl;
      for (int i = 0; i < h_cuts_ptr.size(); i++) {
        std::cout << h_cuts_ptr[i] << " ";
      }
      std::cout << std::endl;

      // Get bin_idx matrix



      //common::Span<std::int32_t const> bin_idx
      //plugin->Reset(h_cuts_ptr, bin_idx);
      is_aggr_context_initialized_ = true;
    }

    std::cout << "Transmitting row indices to plugin" << std::endl;
    // print a few samples of ridx
    std::vector<uint32_t> h_ridx(ridx.size());
    dh::CopyDeviceSpanToVector(&h_ridx, ridx);
    std::cout << "ridx.size() = " << h_ridx.size() << std::endl;
    for (int i = 0; i < 5; i++) {
      std::cout << h_ridx[i] << " ";
    }
    std::cout << std::endl;

    // Transmit row indices to plugin and get encrypted histogram
    //hist_data_ = this->plugin_->BuildEncryptedHistVert(ptrs, sizes, nodes);

    // Perform AllGather
    std::cout << "Allgather histograms" << std::endl;
    /*
    HostDeviceVector<std::int8_t> hist_entries;
    std::vector<std::int64_t> recv_segments;
    collective::SafeColl(
              collective::AllgatherV(ctx_, linalg::MakeVec(hist_data_), &recv_segments, &hist_entries));

    // Call the plugin here to get the resulting histogram. Histogram from all workers are
    // gathered to the label owner.
    common::Span<double> hist_aggr =
            plugin_->SyncEncryptedHistVert(common::RestoreType<std::uint8_t>(hist_entries.HostSpan()));
*/







    // !!!Temporarily turn on regular histogram building for testing
    // compute local histograms
    this->p_impl_->BuildHistogram(ctx->CUDACtx(), matrix, feature_groups, gpair, ridx, histogram, rounding);

    // Further histogram sync process - simulated with allreduce
    // copy histogram data to host
    std::vector<GradientPairInt64> host_histogram(histogram.size());
    dh::CopyDeviceSpanToVector(&host_histogram, histogram);
    // convert to regular vector
    std::vector<std::int64_t> host_histogram_64(histogram.size() * 2);
    for (auto i = 0; i < host_histogram.size(); i++) {
        host_histogram_64[i * 2] = host_histogram[i].GetQuantisedGrad();
        host_histogram_64[i * 2 + 1] = host_histogram[i].GetQuantisedHess();
    }
    // aggregate histograms in float
    auto rc = collective::Allreduce(ctx, &host_histogram_64, collective::Op::kSum);
    SafeColl(rc);
    // convert back to GradientPairInt64
    // only copy to Rank 0, clear other ranks to simulate the plugin scenario
    for (auto i = 0; i < host_histogram.size(); i++) {
      GradientPairInt64 hist_item(host_histogram_64[i * 2], host_histogram_64[i * 2 + 1]);
      GradientPairInt64 hist_item_empty(0, 0);
      if (collective::GetRank() != 0) {
        hist_item = hist_item_empty;
      } else {
        host_histogram[i] = hist_item;
      }
    }
    // copy the aggregated histogram back to GPU memory
    // at this point, the histogram contains full information from all parties
    dh::safe_cuda(hipMemcpyAsync(histogram.data(), host_histogram.data(),
                                  histogram.size() * sizeof(GradientPairPrecise),
                                  hipMemcpyHostToDevice));

  }
}
}  // namespace xgboost::tree
